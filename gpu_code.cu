#include "hip/hip_runtime.h"
 #include <hip/hip_runtime.h>

#include<stdio.h>
#include<stdlib.h>


#include "petya.h"
#include "keyCandidateDistributor.h"

#include <boost/thread.hpp>
#include <boost/container/vector.hpp>

#include "globals.h"
#include "gpu_code.h"

#define NR_THREADS 1024
#define NR_BLOCKS 1
#define NR_OF_KEYS_CALCULATED_BEFORE_THREAD_RETURNS (uint64_t)10000
#define NR_KEYS_PER_GPU_CALL (uint64_t)(NR_THREADS*NR_BLOCKS)


// Define this to turn on error checking
#define CUDA_ERROR_CHECK

#define CudaSafeCall( err ) __cudaSafeCall( err, __FILE__, __LINE__ )
#define CudaCheckError()    __cudaCheckError( __FILE__, __LINE__ )

inline void __cudaSafeCall( hipError_t err, const char *file, const int line )
{
#ifdef CUDA_ERROR_CHECK
    if ( hipSuccess != err )
    {
        fprintf( stderr, "cudaSafeCall() failed at %s:%i : %s\n",
                 file, line, hipGetErrorString( err ) );
        exit( -1 );
    }
#endif

    return;
}

inline void __cudaCheckError( const char *file, const int line )
{
#ifdef CUDA_ERROR_CHECK
    hipError_t err = hipGetLastError();
    if ( hipSuccess != err )
    {
        fprintf( stderr, "cudaCheckError() failed at %s:%i : %s\n",
                 file, line, hipGetErrorString( err ) );
        exit( -1 );
    }

    // More careful checking. However, this will affect performance.
    // Comment away if needed.
    err = hipDeviceSynchronize();
    if( hipSuccess != err )
    {
        fprintf( stderr, "cudaCheckError() with sync failed at %s:%i : %s\n",
                 file, line, hipGetErrorString( err ) );
        exit( -1 );
    }
#endif

    return;
}


using namespace std;


/*
__device__ void calculateSingleShot(uint8_t *keys,
        
        uint8_t nonce[8],
        uint8_t *buf,
        uint32_t buflen,
        bool *isValid,
		int nrTotal) {
	
}*/

__global__ void gpu_decryptSingleShot(uint8_t *keys,
                           
                            uint8_t nonce[8],
                            uint8_t *buf,
                            uint32_t buflen,
                            bool *isValid,
							int nrTotal)
{
	
  int threadNr = blockDim.x * blockIdx.x + threadIdx.x;

  if (threadNr>=nrTotal) return;
  
  uint8_t *key = keys + (threadNr*(KEY_SIZE));   
  
  (isValid)[threadNr] = false;

  uint8_t keystream[64];
  uint8_t n[16] = { 0 };
  uint32_t i;

  for (i = 0; i < 8; ++i)
	n[i] = nonce[i];

  uint8_t *validationBuffer;
  
  validationBuffer = buf + (threadNr*(KEY_SIZE));
  
  for (int bufPos = 0; bufPos < buflen; ++bufPos) {
	  
	  
	if (bufPos % 64 == 0) {
	  //s20_rev_littleendian(n+8, ((si + i) / 64));
	  (n+8)[0] = (bufPos / 64);
	  (n+8)[1] = (bufPos / 64)>>8;
	  (n+8)[2] = (bufPos / 64)>>16;
	  (n+8)[3] = (bufPos / 64)>>24;

	  // s20_expand16(key, n, keystream);

	  int i, j;
	  uint8_t t[4][4] = {
		{ 'e', 'x', 'p', 'a' },
		{ 'n', 'd', ' ', '1' },
		{ '6', '-', 'b', 'y' },
		{ 't', 'e', ' ', 'k' }
	  };

	  for (i = 0; i < 64; i += 20)
		for (j = 0; j < 4; ++j)
		  keystream[i + j] = t[i / 20][j];

	  for (i = 0; i < 16; ++i) {
		keystream[4+i]  = key[i];
		keystream[44+i] = key[i];
		keystream[24+i] = n[i];
	  }

  // ____________________
  // s20_hash(keystream);
  // --------------------

	//    int i;
	  uint32_t x[16];
	  uint32_t z[16];

	  for (i = 0; i < 16; ++i) {

		// s20_littleendian
		uint8_t* result = keystream + (4 * i);
		x[i] = z[i] = (int16_t)(result[0]+(result[1]<<8)); //  s20_littleendian(seq + (4 * i));
	  }

	  for (i = 0; i < 10; ++i) {
			//    s20_doubleround(z);

			  // ColumnRound
			  // s20_quarterround(&x[0], &x[4], &x[8], &x[12]);

			  z[4] =  z[4]  ^ ROTL(z[0]  + z[12], 7);
			  z[8] =  z[8]  ^ ROTL(z[4]  + z[0], 9);
			  z[12] = z[12] ^ ROTL(z[8]  + z[4], 13);
			  z[0] =  z[0]  ^ ROTL(z[12] + z[8], 18);

			  // s20_quarterround(&x[5], &x[9], &x[13], &x[1]);
			  z[9] =  z[9]  ^ ROTL(z[5]  + z[1], 7);
			  z[13] = z[13] ^ ROTL(z[9]  + z[5], 9);
			  z[1] =  z[1]  ^ ROTL(z[13] + z[9], 13);
			  z[5] =  z[5]  ^ ROTL(z[1]  + z[13], 18);

			  // s20_quarterround(&x[10], &x[14], &x[2], &x[6]);
			  z[14]=  z[14] ^ ROTL(z[10] + z[6], 7);
			  z[2] =  z[2]  ^ ROTL(z[14] + z[10], 9);
			  z[6] =  z[6]  ^ ROTL(z[2]  + z[14], 13);
			  z[10] = z[10] ^ ROTL(z[6]  + z[2], 18);

			  // s20_quarterround(&x[15], &x[3], &x[7], &x[11]);
			  z[3] =  z[3]  ^ ROTL(z[15] + z[11], 7);
			  z[7] =  z[7]  ^ ROTL(z[3]  + z[15], 9);
			  z[11] = z[11] ^ ROTL(z[7]  + z[3], 13);
			  z[15] = z[15] ^ ROTL(z[11] + z[7], 18);

			  // Rowround
			  // s20_quarterround(&y[0], &y[1], &y[2], &y[3]);
			  z[1] = z[1] ^ ROTL(z[0]+  z[3], 7);
			  z[2] = z[2] ^ ROTL(z[1] + z[0], 9);
			  z[3] = z[3] ^ ROTL(z[2] + z[1], 13);
			  z[0] = z[0] ^ ROTL(z[3] + z[2], 18);

			  // s20_quarterround(&y[5], &y[6], &y[7], &y[4]);
			  z[6] = z[6] ^ ROTL(z[5] + z[4], 7);
			  z[7] = z[7] ^ ROTL(z[6] + z[5], 9);
			  z[4] = z[4] ^ ROTL(z[7] + z[6], 13);
			  z[5] = z[5] ^ ROTL(z[4] + z[7], 18);

			  // s20_quarterround(&y[10], &y[11], &y[8], &y[9]);
			  z[11] = z[11] ^ ROTL(z[10] + z[9], 7);
			  z[8] =  z[8]  ^ ROTL(z[11] + z[10], 9);
			  z[9] =  z[9]  ^ ROTL(z[8] +  z[11], 13);
			  z[10] = z[10] ^ ROTL(z[9] +  z[8], 18);

			  // s20_quarterround(&y[15], &y[12], &y[13], &y[14]);
			  z[12] = z[12] ^ ROTL(z[15] + z[14], 7);
			  z[13] = z[13] ^ ROTL(z[12] + z[15], 9);
			  z[14] = z[14] ^ ROTL(z[12] + z[13], 13);
			  z[15] = z[15] ^ ROTL(z[14] + z[13], 18);
		  }

		  for (i = 0; i < 16; ++i) {
			z[i] += x[i];
			// s20_rev_littleendian(seq + (4 * i), z[i]);
			  (keystream + (4 * i))[0] = z[i];
			  (keystream + (4 * i))[1] = z[i] >> 8;
			  (keystream + (4 * i))[2] = z[i] >> 16;
			  (keystream + (4 * i))[3] = z[i] >> 24;
		  }

		}
			validationBuffer[bufPos] ^= keystream[ bufPos % 64];
	  }

	  (isValid)[threadNr] = true; // Assume we found the key

	  
	  // Validate Crypto Result
	  for (size_t bufPos = 0; bufPos < VERIBUF_SIZE; bufPos++) {
		 if (validationBuffer[bufPos] != VERIFICATION_CHAR) {
			(isValid)[threadNr] = false; // We didn't								
			break;
		}
		
	  }
	  
  }



__global__ void gpu_decryptMultiShot(uint8_t *keys,                           
                            uint8_t nonce[8],
                            uint8_t *buf,
                            uint32_t buflen,
                            bool *isValid,
							int nrTotal,
							uint64_t nrKeysToCalculatePerThreadBeforeReturn,
							char *keyChars,
							int *keyToIndexMap)
{
	  int threadNr = blockDim.x * blockIdx.x + threadIdx.x;
	  
	  if (threadNr>=nrTotal) return;
	  ///
	  uint8_t keystreamCopy[64];
	  uint8_t nCopy[16] = { 0 };
	  uint32_t i;

	  for (i = 0; i < 8; ++i)
		nCopy[i] = nonce[i];


//		if (bufPos % 64 == 0) {
		  //s20_rev_littleendian(n+8, ((si + i) / 64));
		  (nCopy+8)[0] = 0;// (bufPos / 64);
		  (nCopy+8)[1] = 0;//(bufPos / 64)>>8;
		  (nCopy+8)[2] = 0;//(bufPos / 64)>>16;
		  (nCopy+8)[3] = 0;//(bufPos / 64)>>24;

		  // s20_expand16(key, n, keystream);

		  //int i; 
		  int j;
		  uint8_t t[4][4] = {
			{ 'e', 'x', 'p', 'a' },
			{ 'n', 'd', ' ', '1' },
			{ '6', '-', 'b', 'y' },
			{ 't', 'e', ' ', 'k' }
		  };

		  for (i = 0; i < 64; i += 20)
			for (j = 0; j < 4; ++j)
			  keystreamCopy[i + j] = t[i / 20][j];
		  
	  ///
	  
	  bool keyFound = false;

	  uint8_t *key = keys + (threadNr*(KEY_SIZE));   
	  
	  while (nrKeysToCalculatePerThreadBeforeReturn>0 && !keyFound) 
	  {
		  (isValid)[threadNr+1] = false;

		  
		  uint8_t keystream[64];
		  uint8_t n[16];// = { 0 };
		  uint32_t i;

		 // hipMemcpy(n, nCopy, 16, hipMemcpyDeviceToDevice);
		 // hipMemcpy(keystream, keystreamCopy, 64, hipMemcpyDeviceToDevice);

		  for (int i=0; i<16; i++) {
			  n[i] = nCopy[i];
			  keystream[i] = keystreamCopy[i];
		  }
		  for (int i=16; i<64;i++) {
			  keystream[i] = keystreamCopy[i];			  
		  }
		  
		  uint8_t *validationBuffer;
		  
		  validationBuffer = buf + (threadNr*(KEY_SIZE));


		  for (i = 0; i < 16; ++i) {
			keystream[4+i]  = key[i];
			keystream[44+i] = key[i];
			keystream[24+i] = n[i];
		  }

		  // ____________________
		  // s20_hash(keystream);
		  // --------------------

			//    int i;
			  uint32_t x[16];
			  uint32_t z[16];

			  for (i = 0; i < 16; ++i) {

				// s20_littleendian
				uint8_t* result = keystream + (4 * i);
				x[i] = z[i] = (int16_t)(result[0]+(result[1]<<8)); //  s20_littleendian(seq + (4 * i));
			  }

			  for (i = 0; i < 10; ++i) {
					//    s20_doubleround(z);

					  // ColumnRound
					  // s20_quarterround(&x[0], &x[4], &x[8], &x[12]);

					  z[4] =  z[4]  ^ ROTL(z[0]  + z[12], 7);
					  z[8] =  z[8]  ^ ROTL(z[4]  + z[0],  9);
					  z[12] = z[12] ^ ROTL(z[8]  + z[4], 13);
					  z[0] =  z[0]  ^ ROTL(z[12] + z[8], 18);

					  // s20_quarterround(&x[5], &x[9], &x[13], &x[1]);
					  z[9] =  z[9]  ^ ROTL(z[5]  + z[1],   7);
					  z[13] = z[13] ^ ROTL(z[9]  + z[5],   9);
					  z[1] =  z[1]  ^ ROTL(z[13] + z[9],  13);
					  z[5] =  z[5]  ^ ROTL(z[1]  + z[13], 18);

					  // s20_quarterround(&x[10], &x[14], &x[2], &x[6]);
					  z[14]=  z[14] ^ ROTL(z[10] + z[6],   7);
					  z[2] =  z[2]  ^ ROTL(z[14] + z[10],  9);
					  z[6] =  z[6]  ^ ROTL(z[2]  + z[14], 13);
					  z[10] = z[10] ^ ROTL(z[6]  + z[2],  18);

					  // s20_quarterround(&x[15], &x[3], &x[7], &x[11]);
					  z[3] =  z[3]  ^ ROTL(z[15] + z[11], 7);
					  z[7] =  z[7]  ^ ROTL(z[3]  + z[15], 9);
					  z[11] = z[11] ^ ROTL(z[7]  + z[3], 13);
					  z[15] = z[15] ^ ROTL(z[11] + z[7], 18);

					  // Rowround
					  // s20_quarterround(&y[0], &y[1], &y[2], &y[3]);
					  z[1] = z[1] ^ ROTL(z[0]+  z[3], 7);
					  z[2] = z[2] ^ ROTL(z[1] + z[0], 9);
					  z[3] = z[3] ^ ROTL(z[2] + z[1], 13);
					  z[0] = z[0] ^ ROTL(z[3] + z[2], 18);

					  // s20_quarterround(&y[5], &y[6], &y[7], &y[4]);
					  z[6] = z[6] ^ ROTL(z[5] + z[4], 7);
					  z[7] = z[7] ^ ROTL(z[6] + z[5], 9);
					  z[4] = z[4] ^ ROTL(z[7] + z[6], 13);
					  z[5] = z[5] ^ ROTL(z[4] + z[7], 18);

					  // s20_quarterround(&y[10], &y[11], &y[8], &y[9]);
					  z[11] = z[11] ^ ROTL(z[10] + z[9], 7);
					  z[8] =  z[8]  ^ ROTL(z[11] + z[10], 9);
					  z[9] =  z[9]  ^ ROTL(z[8] +  z[11], 13);
					  z[10] = z[10] ^ ROTL(z[9] +  z[8], 18);

					  // s20_quarterround(&y[15], &y[12], &y[13], &y[14]);
					  z[12] = z[12] ^ ROTL(z[15] + z[14], 7);
					  z[13] = z[13] ^ ROTL(z[12] + z[15], 9);
					  z[14] = z[14] ^ ROTL(z[12] + z[13], 13);
					  z[15] = z[15] ^ ROTL(z[14] + z[13], 18);
				  }

				  for (i = 0; i < 16; ++i) {
					z[i] += x[i];
					// s20_rev_littleendian(seq + (4 * i), z[i]);
					  (keystream + (4 * i))[0] = z[i];
					  (keystream + (4 * i))[1] = z[i] >> 8;
					  (keystream + (4 * i))[2] = z[i] >> 16;
					  (keystream + (4 * i))[3] = z[i] >> 24;
				  }

//				}

		  
	  	  (isValid)[threadNr+1] = true; // Assume we found the key

	 
		  for (int bufPos = 0; bufPos < buflen; ++bufPos) {
			  
			  
					char c = validationBuffer[bufPos];
					c ^= keystream[ bufPos]; // % 64 
					if (c!=VERIFICATION_CHAR) {
						(isValid)[threadNr+1] = false;
						
						// Calculate next key to try...
						int posToKey[] = {13,12,9,8,5,4,1,0};

						for (int i=0; i<8; i++) {
							int idx = keyToIndexMap[(char)key[posToKey[i]]];
							idx++;
							idx %= (2 * 26 + 10);
							key[posToKey[i]] = keyChars[idx];

							if (idx!=0) break;
						}				
						break;
					}
			  }
		  
		  	  
	

			  if ((isValid)[threadNr+1]==true) {
				  keyFound = true;
				  (isValid)[0] = true; // set first index to true to inducate key was found in one of the threads 
			  }
			  
			  
			  
			  nrKeysToCalculatePerThreadBeforeReturn--;
	  	  }
	  	  
	  	  

}


void tryKeysGPUSingleShot(uint64_t nrBlocks,
				uint64_t nrThreads,
				uint8_t nonce_hc[8],  
		        char *verificationBuffer, 
				char*keys, 
				uint64_t nrKeys, 
				bool *result) {
    
    uint8_t *verificationBuffer_hc;
    uint8_t *verifbuf_test_dc = NULL;
    uint8_t *keys_dc;                       
    uint8_t *nonce_dc;
    bool *result_dc;
    
    verificationBuffer_hc = (uint8_t *) malloc(VERIBUF_SIZE*nrKeys);
    
    // Fill verificationBuffer for each thread...
    for (uint64_t i=0; i<nrKeys; i++) {
    	memcpy(verificationBuffer_hc+i*KEY_SIZE, verificationBuffer, VERIBUF_SIZE);
    }
                    
    CudaSafeCall(hipMalloc((void **)&verifbuf_test_dc, (VERIBUF_SIZE*nrKeys)));
    CudaSafeCall(hipMemcpy(verifbuf_test_dc, verificationBuffer_hc, (VERIBUF_SIZE*nrKeys), hipMemcpyHostToDevice));
    CudaSafeCall(hipMalloc((void **)&keys_dc, (KEY_SIZE)*nrKeys));
    CudaSafeCall(hipMalloc((void **)&nonce_dc, 8));
    CudaSafeCall(hipMemcpy(nonce_dc, nonce_hc, 8, hipMemcpyHostToDevice));
    CudaSafeCall(hipMalloc((void **)&result_dc, sizeof(bool)*nrKeys));
    

        
    CudaSafeCall(hipMemcpy(keys_dc, (uint8_t *) keys, (KEY_SIZE)*nrKeys, hipMemcpyHostToDevice));

    gpu_decryptSingleShot<<<nrBlocks, nrThreads>>>(keys_dc, 
                                         nonce_dc, 
                                         verifbuf_test_dc, 
                                         VERIBUF_SIZE, 
                                         result_dc,nrKeys);
    CudaCheckError();
        
    CudaSafeCall(hipMemcpy(result, result_dc, sizeof(bool)*nrKeys, hipMemcpyDeviceToHost));        
    CudaSafeCall(hipMemcpy((uint8_t *) keys, keys_dc, (KEY_SIZE)*nrKeys, hipMemcpyDeviceToHost));
        
       
	free(verificationBuffer_hc);

	// Free device global memory
	CudaSafeCall(hipFree(result_dc));    
	CudaSafeCall(hipFree(verifbuf_test_dc));    
}

void printKeys(char*keys, uint64_t nrKeys) {
	char* currentKey = keys;
	uint64_t prevKeyIdx = 0;

	for (unsigned long i = 0; i < nrKeys; i++){
		uint64_t keyIdx = calculateIndexFrom16ByteKey(currentKey);
		for (int j=0; j<KEY_SIZE; j++) {
			printf("%c",currentKey[j]);
		}
		cout << " "<< (keyIdx);
		if (keyIdx>prevKeyIdx)
		cout << " "<< (keyIdx-prevKeyIdx) << endl;
		else
		cout << " "<< (prevKeyIdx-keyIdx) << endl;
			
		prevKeyIdx = keyIdx;
		currentKey += KEY_SIZE;
	}			
	cout << endl;
}

// Wrapper for Boost::Thread

bool tryKeysGPUMultiShot(const GPUMultiShotArguments &argument) {
		
	GPUMultiShotArguments arguments2 = argument;
	
 	return tryKeysGPUMultiShot(arguments2.nrBlocks,
			arguments2.nrThreads,
			arguments2.nonce_hc,
			arguments2.verificationBuffer,
			arguments2.keys,
			arguments2.nrKeys,
			arguments2.keysBeforeContextSwitch,
			arguments2.keysInTotalToCalculate,
			arguments2.supressOutput,
			arguments2.shutdownRequested);			
}


bool tryKeysGPUMultiShot(uint64_t nrBlocks,
				uint64_t nrThreads,
				uint8_t nonce_hc[8],  
		        char *verificationBuffer, 
				char*keys, 
				uint64_t nrKeys,
				uint64_t keysBeforeContextSwitch,
				uint64_t keysInTotalToCalculate,
				bool supressOutput,
				bool* shutdownRequested) {
    

	uint64_t nrTotalKeys = pow(26*2+10,8);
	
    uint8_t *verificationBuffer_hc;
    uint8_t *verifbuf_test_dc = NULL;
    uint8_t *keys_dc;                       
    uint8_t *nonce_dc;
    bool *result_dc;
    
    char *keyChars_dc;
    int *keyToIndexMap_dc;

    bool *result = (bool *)malloc(sizeof(bool)*(nrKeys+1)); //+1 as 0 Index stores information if key was found by one thread at all
    
    memset(result, 0, sizeof(bool)*(nrKeys+1));

    
    // This is initialized here to save time to calculate next key later, todo: Calculate outside and provide with parameters...
    char keyChars[] = "0123456789abcdefghijklmnopqrstuvwxyzABCDEFGHIJKLMNOPQRSTUVWXYZ";
     
    int keyToIndexMap[256];
    for (int i=0; i<sizeof(keyChars);i++) {
  	  for (int j=0; j<256;j++) {
  		  if (keyChars[i]==(char)j) {
  			  keyToIndexMap[(char)j]=i;
  		  }
  	  }
    }
              
    
    verificationBuffer_hc = (uint8_t *) malloc(VERIBUF_SIZE*nrKeys);
    
    // Fill verificationBuffer for each thread...
    for (uint64_t i=0; i<nrKeys; i++) {
    	memcpy(verificationBuffer_hc+i*KEY_SIZE, verificationBuffer, VERIBUF_SIZE);
    }
                    
    CudaSafeCall(hipMalloc((void **)&verifbuf_test_dc, (VERIBUF_SIZE*nrKeys)));
    CudaSafeCall(hipMemcpy(verifbuf_test_dc, verificationBuffer_hc, (VERIBUF_SIZE*nrKeys), hipMemcpyHostToDevice));
    CudaSafeCall(hipMalloc((void **)&keys_dc, (KEY_SIZE)*nrKeys));
    CudaSafeCall(hipMalloc((void **)&nonce_dc, 8));
    CudaSafeCall(hipMemcpy(nonce_dc, nonce_hc, 8, hipMemcpyHostToDevice));
    CudaSafeCall(hipMalloc((void **)&result_dc, sizeof(bool)*(nrKeys+1))); // +1 as 0 Index stores information if key was found by one thread at all
    CudaSafeCall(hipMemcpy(result_dc, result, sizeof(bool)*(nrKeys+1), hipMemcpyHostToDevice));
    

    CudaSafeCall(hipMalloc((void **)&keyChars_dc, sizeof(keyChars)));
    CudaSafeCall(hipMemcpy(keyChars_dc, keyChars, sizeof(keyChars), hipMemcpyHostToDevice));
    CudaSafeCall(hipMalloc((void **)&keyToIndexMap_dc, 256*sizeof(int)));
    CudaSafeCall(hipMemcpy(keyToIndexMap_dc, keyToIndexMap, 256*sizeof(int), hipMemcpyHostToDevice));

    
    bool keyFound = false;
    uint64_t keysCalculated = 0;
    
    boost::posix_time::time_duration duration;
    boost::posix_time::ptime beginTs = boost::posix_time::second_clock::local_time();

    CudaSafeCall(hipMemcpy(keys_dc, (uint8_t *) keys, (KEY_SIZE)*nrKeys, hipMemcpyHostToDevice));

    /*
    for (int i=0; i<(KEY_SIZE)*nrKeys; i++) {
    	printf("%c",keys[i]);
    }
    cout << endl;
    */
    
    // printKeys(keys, nrKeys);
    //cout << "Now starting calculation"<<endl;
    
	int lastPrintedPercentRange = -1;
	int lastPrintedPercentTotal = -1;


    do {
    
		gpu_decryptMultiShot<<<nrBlocks, nrThreads>>>(keys_dc, 
											 nonce_dc, 
											 verifbuf_test_dc, 
											 VERIBUF_SIZE, 
											 result_dc,nrKeys,
											 keysBeforeContextSwitch,
											 keyChars_dc,
											 keyToIndexMap_dc);


		CudaCheckError();


		CudaSafeCall(hipMemcpy(result, result_dc, sizeof(bool)*(nrKeys+1), hipMemcpyDeviceToHost));        
	
		if (result[0]==true) { // If key was found at all...
			CudaSafeCall(hipMemcpy((uint8_t *) keys, keys_dc, (KEY_SIZE)*nrKeys, hipMemcpyDeviceToHost));
			

			for (int i=1; i<nrKeys+1;i++) {
				if (result[i]) {

					if (!supressOutput) {
						printf("Key found:\r\n");
						for (int j = 0; j < KEY_SIZE; j++) {
							printf("%c", keys[(KEY_SIZE)*(i - 1) + j]); // -1 as 0 index is reserved to store if result was found at all
						}
						printf("\r\n");
					}
					keyFound = true;
				}
			}
		}
	

		// Keys for next round should have been already calculated on GPU
		
		// Calculate next keys for next round...
		// for (uint64_t i=0;i<NR_KEYS_PER_GPU_CALL;i++) {
		//	char *currentKey = (key+i*KEY_SIZE); 
		//	nextKey16Byte(currentKey);
		// }
		
		keysCalculated += (uint64_t)nrThreads*(uint64_t)nrBlocks*keysBeforeContextSwitch;
		
// 		if (keysCalculated%1000000 == 0) {
			int currentRange = (int)((uint64_t) (keysCalculated * (uint64_t)100 / keysInTotalToCalculate));
			int currentTotal = (int)((uint64_t)keysCalculated * (uint64_t)100 / nrTotalKeys);

			if (currentRange > 100) currentRange = 100;
			if (currentTotal > 100) currentTotal = 100;


			if (lastPrintedPercentRange != currentRange || currentTotal != lastPrintedPercentTotal){
				lastPrintedPercentRange = currentRange;
				lastPrintedPercentTotal = currentTotal;

				if (!supressOutput) cout << lastPrintedPercentRange << "% of Job calculated, that's " << lastPrintedPercentTotal << "% of the whole key range" << endl;
			}
//		}
		    
    } while (!keyFound &&  keysCalculated<keysInTotalToCalculate && !(*shutdownRequested));

	if (*shutdownRequested){
		CudaSafeCall(hipMemcpy((uint8_t *)keys, keys_dc, (KEY_SIZE)*nrKeys, hipMemcpyDeviceToHost));
		// printKeys(keys, nrKeys);
	}

	// Free device global memory

	CudaSafeCall(hipFree(keyChars_dc));
	CudaSafeCall(hipFree(keyToIndexMap_dc));    

	
	CudaSafeCall(hipFree(keys_dc));
	CudaSafeCall(hipFree(result_dc));    
	CudaSafeCall(hipFree(nonce_dc));    
	CudaSafeCall(hipFree(verifbuf_test_dc));  

	free(verificationBuffer_hc);
	free(result);

	cout << (*shutdownRequested) << endl;
	cout<< keysCalculated << endl;
	cout<< keysInTotalToCalculate << endl;

	cout << "ending number crunching"<<endl;

	return keyFound;
}


void measureGPUPerformance(uint64_t nrBlocks,
				uint64_t nrThreads, 
				uint64_t keysBeforeContextSwitch, 
				uint64_t *nrKeysCalculatedResult,
				uint64_t *nrOfSecondsInTotalMeasured,
				bool* shutdownRequested,
				int nrSecondsToMeasure) {
    
    uint8_t *verificationBuffer_hc;
    uint8_t *verifbuf_test_dc = NULL;
    uint8_t *keys_dc;                       
    uint8_t *nonce_dc;
    bool *result_dc;
    
    char *keyChars_dc;
    int *keyToIndexMap_dc;

    
    uint64_t nrKeys = nrThreads * nrBlocks;
    bool *result = (bool *)malloc(sizeof(bool)*(nrKeys+1));
    char*keys = (char*) malloc(sizeof(char)*nrKeys*KEY_SIZE);
    
    memset(result, 0, sizeof(bool)*(nrKeys+1));
    
    	
    uint64_t keysCalculated = 0;

	uint8_t nonce_hc[8];
    char *verificationBuffer = (char *)malloc(VERIBUF_SIZE);
	
	
	nonce_hc[0] = 0x07;
	nonce_hc[1] = 0x0c;
	nonce_hc[2] = 0x12;
	nonce_hc[3] = 0xf6;
	nonce_hc[4] = 0x79;
	nonce_hc[5] = 0x28;
	nonce_hc[6] = 0x73;
	nonce_hc[7] = 0xcb;

	verificationBuffer[0] = 0x34;
	verificationBuffer[1] = 0x80;
	verificationBuffer[2] = 0x15;
	verificationBuffer[3] = 0x1a;
	verificationBuffer[4] = 0xd1;
	verificationBuffer[5] = 0x76;
	verificationBuffer[6] = 0x5c;
	verificationBuffer[7] = 0x7b;
	verificationBuffer[8] = 0x60;
	verificationBuffer[9] = 0x2b;
	verificationBuffer[10] = 0xe3;
	verificationBuffer[11] = 0xd0;
	verificationBuffer[12] = 0xd0;
	verificationBuffer[13] = 0xae;
	verificationBuffer[14] = 0xf8;
	verificationBuffer[15] = 0xc2;
	
    // This is initialized here to save time to calculate next key later, todo: Calculate outside and provide with parameters...
    char keyChars[] = "0123456789abcdefghijklmnopqrstuvwxyzABCDEFGHIJKLMNOPQRSTUVWXYZ";
      
    int keyToIndexMap[256];
    for (int i=0; i<sizeof(keyChars);i++) {
  	  for (int j=0; j<256;j++) {
  		  if (keyChars[i]==(char)j) {
  			  keyToIndexMap[(char)j]=i;
  		  }
  	  }
    }

    verificationBuffer_hc = (uint8_t *) malloc(VERIBUF_SIZE*nrKeys);
   
    
	
    // Fill verificationBuffer for each thread...
    for (uint64_t i=0; i<nrKeys; i++) {
    	memcpy(verificationBuffer_hc+i*KEY_SIZE, verificationBuffer, VERIBUF_SIZE);
    }
    
    
	// memset(keys,'0', nrKeys*KEY_SIZE);
    uint64_t keyBlocks = pow(26*2+10,8)/(nrKeys);
    
    char *currentKey = keys;
    for (uint64_t i=0; i<nrKeys;i++){
    	calculate16ByteKeyFromIndex(0+i*keyBlocks, currentKey);
    	currentKey+=KEY_SIZE;
    }
    


    

    
    CudaSafeCall(hipMalloc((void **)&verifbuf_test_dc, (VERIBUF_SIZE*nrKeys)));
    CudaSafeCall(hipMemcpy(verifbuf_test_dc, verificationBuffer_hc, (VERIBUF_SIZE*nrKeys), hipMemcpyHostToDevice));
    CudaSafeCall(hipMalloc((void **)&nonce_dc, 8));
    CudaSafeCall(hipMemcpy(nonce_dc, nonce_hc, 8, hipMemcpyHostToDevice));
    CudaSafeCall(hipMalloc((void **)&result_dc, sizeof(bool)*(nrKeys+1)));
    CudaSafeCall(hipMemcpy(result_dc, result, sizeof(bool)*(nrKeys+1), hipMemcpyHostToDevice));
    CudaSafeCall(hipMalloc((void **)&keys_dc, sizeof(uint8_t)*(KEY_SIZE)*nrKeys));

    CudaSafeCall(hipMalloc((void **)&keyChars_dc, sizeof(keyChars)));
    CudaSafeCall(hipMemcpy(keyChars_dc, keyChars, sizeof(keyChars), hipMemcpyHostToDevice));
    CudaSafeCall(hipMalloc((void **)&keyToIndexMap_dc, 256*sizeof(int)));
    CudaSafeCall(hipMemcpy(keyToIndexMap_dc, keyToIndexMap, 256*sizeof(int), hipMemcpyHostToDevice));



	
    boost::posix_time::time_duration duration;
    boost::posix_time::ptime beginTs = boost::posix_time::second_clock::local_time();


	CudaSafeCall(hipMemcpy(keys_dc, (uint8_t *) keys, (KEY_SIZE)*nrKeys, hipMemcpyHostToDevice));

	do {


		
	

		
		gpu_decryptMultiShot<<<nrBlocks, nrThreads>>>(keys_dc, 
											 nonce_dc, 
											 verifbuf_test_dc, 
											 VERIBUF_SIZE, 
											 result_dc,
											 nrKeys,
											 keysBeforeContextSwitch,
											 keyChars_dc,
											 keyToIndexMap_dc);
		
		CudaCheckError();
		
			
		CudaSafeCall(hipMemcpy(result, result_dc, sizeof(bool)*(nrKeys+1), hipMemcpyDeviceToHost));        


		
		if (result[0]==true) {
			
			CudaSafeCall(hipMemcpy((uint8_t *) keys, keys_dc, (KEY_SIZE)*nrKeys, hipMemcpyDeviceToHost));
			
			cout << endl;
			for (int i=1; i<nrKeys+1;i++) {
				if (result[i]) {
					printf("Key found:\r\n");
					for (int j=0; j<KEY_SIZE; j++) {
						printf("%c", keys[(KEY_SIZE)*(i-1)+j]);
					}
					printf("\r\n");
				}
			}
			
		}
	
		
		keysCalculated += nrThreads*nrBlocks*keysBeforeContextSwitch;
		
		boost::posix_time::ptime now = boost::posix_time::second_clock::local_time();  
		duration = (now-beginTs);

	} while (!(duration.total_seconds()>nrSecondsToMeasure) && !(*shutdownRequested));

	
	
    
    /*
    // Round up according to array size 
    gridSize = (nrKeys + blockSize - 1) / blockSize; 

	gpu_decryptMultiShot<<<gridSize, blockSize>>>(keys_dc, 
										 nonce_dc, 
										 verifbuf_test_dc, 
										 VERIBUF_SIZE, 
										 result_dc,
										 nrKeys,
										 keysBeforeContextSwitch,
										 keyChars_dc,
										 keyToIndexMap_dc);
	CudaCheckError();
    hipDeviceSynchronize(); 

	// calculate theoretical occupancy
	  int maxActiveBlocks;
	  hipOccupancyMaxActiveBlocksPerMultiprocessor( &maxActiveBlocks, 
			  	  	  	  	  	  	  	  	  	  	 gpu_decryptMultiShot, blockSize, 
	                                                 0);

	  int device;
	  hipDeviceProp_t props;
	  hipGetDevice(&device);
	  hipGetDeviceProperties(&props, device);

	  float occupancy = (maxActiveBlocks * blockSize / props.warpSize) / 
	                    (float)(props.maxThreadsPerMultiProcessor / 
	                            props.warpSize);

	  printf("Launched blocks of size %d. Theoretical occupancy: %f\n", 
	         blockSize, occupancy);
	  
	*/

	
	
	// Free device global memory

	CudaSafeCall(hipFree(keyChars_dc));
	CudaSafeCall(hipFree(keyToIndexMap_dc));    

	
	CudaSafeCall(hipFree(keys_dc));
	CudaSafeCall(hipFree(result_dc));    
	CudaSafeCall(hipFree(nonce_dc));    
	CudaSafeCall(hipFree(verifbuf_test_dc));  

	free(verificationBuffer_hc);
    free(verificationBuffer);
    free(keys);
	free(result);

	*nrOfSecondsInTotalMeasured = duration.total_seconds();
	*nrKeysCalculatedResult = keysCalculated;
}



void queryDeviceInfo(uint64_t* nrOfBlocks, uint64_t* nrThreads) {
	int nDevices;

	  hipGetDeviceCount(&nDevices);
	  for (int i = 0; i < nDevices; i++) {
	    hipDeviceProp_t prop;
	    hipGetDeviceProperties(&prop, i);
	    printf("Device Number: %d\n", i);
	    printf("  Device name: %s\n", prop.name);
	    printf("  Memory Clock Rate (KHz): %d\n",
	           prop.memoryClockRate);
	    printf("  Memory Bus Width (bits): %d\n",
	           prop.memoryBusWidth);
	    printf("  Peak Memory Bandwidth (GB/s): %f\n",
	           2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
//	    printf("  Maximum Threads per Block %d\n", prop.maxThreadsPerBlock);
	    
		// Check again for optimal parameters...
		
	    int blockSize;   // The launch configurator returned block size 
	    int minGridSize; // The minimum grid size needed to achieve the 
	                      // maximum occupancy for a full device launch 
	   //  int gridSize;    // The actual grid size needed, based on input size 

	     
	    hipOccupancyMaxPotentialBlockSize( &minGridSize, &blockSize, 
	    		gpu_decryptMultiShot, 0, 0);
	    
	    cout << "  Recommended BlockSize "<< minGridSize << endl;
	    cout << "  Recommended Threadsize "<< blockSize << endl<<endl;

	    *nrOfBlocks = minGridSize;
	    *nrThreads = blockSize;
	    
	  }
}




